#include "hip/hip_runtime.h"
#include <cutil.h>
#include <stdlib.h>
#include <iostream.h>
#include <cv.h>

const int blocksize = 4;

__global__ void CUDAmakeIntegralImageLignes(uint* a, int width, int height, int pitch ) {
  int n = blockIdx.x * blockDim.x*blockDim.y+blockIdx.y*blockDim.y*blockDim.x*gridDim.x + threadIdx.x + threadIdx.y*blockDim.x;
  uint tmp=0;
  if ( n>=0 && n<height ){
  	for(uint i=0;i<width;i++){
		tmp+= ((uint*)((char*)a + pitch*n))[i];
		((uint*)((char*)a + pitch*n))[i]=tmp;
  	}
  }
}

__global__ void CUDAmakeIntegralImageColonnes(uint* a, int width, int height, int pitch ) {
  int n = blockIdx.x * blockDim.x*blockDim.y+blockIdx.y*blockDim.y*blockDim.x*gridDim.x + threadIdx.x + threadIdx.y*blockDim.x;
  uint tmp=0;
  if ( n>=0 && n<width ){
  	for(uint i=0;i<height;i++){
		tmp+= ((uint*)((char*)a + pitch*i))[n];
		((uint*)((char*)a + pitch*i))[n]=tmp;
  	}
  }
}

void CUDAmakeIntegralImage(const IplImage* in, IplImage* out){
	uint *a, pitch;
	//allocation de la memoire device
	hipMallocPitch((void**)&a,&pitch,in->width*sizeof(unsigned int),in->height);
	
	//copie d'une image U8 vers une S32
	cvConvert(in, out);
	//copie sur le device
	if(hipSuccess != hipMemcpy2D(a,pitch,out->imageData,out->widthStep,
			out->width*sizeof(unsigned int),out->height,hipMemcpyHostToDevice))
			std::cout << "erreur allocation" << std::endl;
	std::cout <<  (int)((uchar*)(in->imageData))[1] << std::endl;
	std::cout <<  ((int*)(out->imageData))[1] << std::endl;
	((int*)(out->imageData))[1]=0;
	//lancement des calculs
	{
	  dim3 dimBlock( blocksize, blocksize );
	  int tmp=(int)(sqrt(in->height/(float)(blocksize*blocksize))+1);
      dim3 dimGrid( tmp,tmp);
      std::cout << "threads pr les lignes " << blocksize*blocksize*tmp*tmp << std::endl;
      CUDAmakeIntegralImageLignes<<<dimGrid, dimBlock>>>( a, in->width, in->height , pitch );
	  hipDeviceSynchronize();
	}
	{
	  dim3 dimBlock( blocksize, blocksize );
	  int tmp=(int)(sqrt(in->width/(float)(blocksize*blocksize))+1);
      dim3 dimGrid( tmp,tmp);
      std::cout << "threads pr les colonnes " <<  blocksize*blocksize*tmp*tmp << std::endl;
      CUDAmakeIntegralImageColonnes<<<dimGrid, dimBlock>>>( a, in->width, in->height , pitch );
	  hipDeviceSynchronize();
	}
	//recuperation depuis le device
	if(hipSuccess != hipMemcpy2D(out->imageData,out->widthStep,a,pitch,
			out->width*sizeof(unsigned int),out->height,hipMemcpyDeviceToHost))
			std::cout << "erreur copie" << std::endl;
	//liberation de la memoire du device
	hipFree(a);
	std::cout << "termine" << std::endl;
	std::cout <<  (int)((uchar*)(in->imageData))[1] << std::endl;
	std::cout <<  ((int*)(out->imageData))[1] << std::endl;
}
