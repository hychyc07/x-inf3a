#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <stdlib.h>
#include <stdio.h>
#include <cv.h>

const int blocksize = 16;

__global__ void CUDAmakeIntegralImageLignes(uint* a, int width, int height, int pitch ) {
  int n = blockIdx.x * blockDim.x*blockDim.y+blockIdx.y*blockDim.y*blockDim.x*gridDim.x + threadIdx.x + threadIdx.y*blockDim.x;
  unsigned int tmp=0;
  if ( n>=0 && n<height ){
  	for(uint i=0;i<width;i++){
		tmp+= ((uint*)(a + pitch*n))[i];
		((uint*)(a + pitch*n))[i]=tmp;
  	}
  }
}

__global__ void CUDAmakeIntegralImageColonnes(uint* a, int width, int height, int pitch ) {
  int n = blockIdx.x * blockDim.x*blockDim.y+blockIdx.y*blockDim.y*blockDim.x*gridDim.x + threadIdx.x + threadIdx.y*blockDim.x;
  unsigned int tmp=0;
  if ( n>=0 && n<width ){
  	for(uint i=0;i<height;i++){
		tmp+= ((uint*)(a + pitch*i))[n];
		((uint*)(a + pitch*i))[n]=tmp;
  	}
  }
}

void CUDAmakeIntegralImage(const IplImage* in, IplImage* out){
	unsigned int *a, pitch;
	//allocation de la memoire device
	hipMallocPitch(&a,&pitch,in->width*sizeof(unsigned int),in->height);
	
	//copie d'une image U8 vers une S32
	cvConvertScale(in, out, 1.);
	//copie sur le device
	hipMemcpy2D(a,pitch,out->imageData,out->widthStep,
			out->width,out->height,hipMemcpyHostToDevice);
	
	//lancement des calculs
	{
	  dim3 dimBlock( blocksize, blocksize );
	  int tmp=(int)(sqrt(in->height/(float)(blocksize*blocksize))+1);
      dim3 dimGrid( tmp,tmp); 
      CUDAmakeIntegralImageLignes<<<dimGrid, dimBlock>>>( a, in->width, in->height , pitch );
	  hipDeviceSynchronize();
	}
	{
	  dim3 dimBlock( blocksize, blocksize );
	  int tmp=(int)(sqrt(in->width/(float)(blocksize*blocksize))+1);
      dim3 dimGrid( tmp,tmp); 
      CUDAmakeIntegralImageColonnes<<<dimGrid, dimBlock>>>( a, in->width, in->height , pitch );
	  hipDeviceSynchronize();
	}
	
	//recuperation du device
	hipMemcpy2D(out->imageData,out->widthStep,a,pitch,
			out->width,out->height,hipMemcpyDeviceToHost);
	//liberation de la memoire du device
	hipFree(a);
}
