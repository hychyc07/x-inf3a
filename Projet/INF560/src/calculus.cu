#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <stdlib.h>
#include <stdio.h>
#include <cv.h>


__global__ void CUDAmakeIntegralImageLignes(float* a, int width, int height, int pitch ) {
  int n = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y; 
  if ( n>=0 && n<height ){
		
	}
}

__global__ void CUDAmakeIntegralImageColonnes(float* a, int width, int height, int pitch ) {

}

void CUDAmakeIntegralImage(const IplImage* in, IplImage* out){
	unsigned int *a, pitch;
	//allocation de la memoire device
	hipMallocPitch(&a,&pitch,in->width*sizeof(unsigned int),in->height);
	
	//copie d'une image U8 vers une S32
	cvConvertScale(in, out, 1.);
	//copie sur le device
	hipMemcpy2D(a,pitch,out->imageData,out->widthStep,
			out->width,out->height,hipMemcpyHostToDevice);
	
	//lancement des calculs
	
	//on attend aue tout le monde ait fini
	hipDeviceSynchronize();
	//recuperation du device
	hipMemcpy2D(out->imageData,out->widthStep,a,pitch,
			out->width,out->height,hipMemcpyDeviceToHost);
	//liberation de la memoire du device
	hipFree(a);
}