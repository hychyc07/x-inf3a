#include <cutil.h>
#include <stdlib.h>
#include <stdio.h>
#include <cv.h>



void CUDAmakeIntegralImage(const IplImage* in, IplImage* out){
	unsigned int *a, pitch;
	//allocation de la memoire device
	hipMallocPitch(&a,&pitch,in->width*4,in->height);
	
	//copie d'une image U8 vers une S32
	cvConvertScale(in, out, 1.);
	//copie sur le device
	hipMemcpy2D(a,pitch,out->imageData,out->widthStep,
			out->width,out->height,hipMemcpyHostToDevice);
	
	//lancement des calculs
	
	//on attend aue tout le monde ait fini
	hipDeviceSynchronize();
	//recuperation du device
	hipMemcpy2D(out->imageData,out->widthStep,a,pitch,
			out->width,out->height,hipMemcpyDeviceToHost);
	//liberation de la memoire du device
	hipFree(a);
}