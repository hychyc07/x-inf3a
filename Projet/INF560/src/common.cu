#ifndef COMMON_H_
#define COMMON_H_

#include <cutil.h>
#include <stdlib.h>
#include <iostream>
#include <cv.h>
#include <ctime>

#include "surfCUDA.h"


void CUDAinit(uint width, uint height, uint intervals){
	//allocation de la memoire device
	if(hipSuccess != hipMallocPitch((void**)&CUDAintegral,&CUDAintegralPitch,width*sizeof(uint),height))
			std::cout << "erreur allocation CUDAintegral" << std::endl;
	if(hipSuccess != hipMallocPitch((void**)&CUDAimg,&CUDAimgPitch,width*sizeof(uint),height))
			std::cout << "erreur allocation CUDAimg" << std::endl;
	CUDAimgs = (uint**) malloc(intervals*sizeof(uint*));
	for(int i=0;i<intervals;i++){
		if(hipSuccess != hipMallocPitch((void**)&(CUDAimgs[i]),&CUDAimgsPitch,width*sizeof(uint),height))
			std::cout << "erreur allocation CUDAimgs " << i << std::endl;
	}
	CUDAadressImgs;
	hipMalloc((void**)&CUDAadressImgs,sizeof(uint*)*intervals);
	hipMemcpy(CUDAadressImgs,CUDAimgs,sizeof(uint*)*intervals,hipMemcpyHostToDevice);
}

void CUDAclose(uint intervals){
	hipFree(CUDAintegral);
	hipFree(CUDAimg);
	hipFree(CUDAadressImgs);
	for(int i=0;i<intervals;i++)
		hipFree(CUDAimgs[i]);
	free(CUDAimgs);
}

#endif
