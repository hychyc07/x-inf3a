#ifndef COMMON_H_
#define COMMON_H_

#include <cutil.h>
#include <stdlib.h>
#include <iostream>
#include <cv.h>
#include <ctime>

#include "surfCUDA.h"


void CUDAinit(uint width, uint height){
	//allocation de la memoire device
	if(hipSuccess != hipMallocPitch((void**)&CUDAintegral,&CUDAintegralPitch,width*sizeof(uint),height))
			std::cout << "erreur allocation CUDAintegral" << std::endl;
	if(hipSuccess != hipMallocPitch((void**)&CUDAimg,&CUDAimgPitch,width*sizeof(uint),height))
			std::cout << "erreur allocation CUDAimg" << std::endl;
}

void CUDAclose(){
	hipFree(CUDAintegral);
	hipFree(CUDAimg);
}

#endif
