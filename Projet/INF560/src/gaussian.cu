#include "hip/hip_runtime.h"
#include <cutil.h>
#include <stdlib.h>
#include <iostream>
#include <cv.h>
#include <ctime>

const int blocksize = 10;

// Acces au pixel x,y d'une image
#define getPixel(in,pitch,x,y) ( ((uint*)((char*)(in) + (pitch)*(x)))[(y)] )

__global__ void CUDAcalculateGaussianDerivative2(uint* integral, uint* out, int width, int height, int pitch , int lobe, int area, int borderSize) {
  int x = blockIdx.x * blockDim.x + threadIdx.x; 
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( x < width-borderSize && y < height-borderSize && x >= borderSize && y >= borderSize ){
  	// Derivee selon x
				int lobeGauche = 0, lobeCentre = 0, lobeDroit = 0 ;
				lobeGauche += getPixel(integral,pitch, x-(lobe+1)/2, y + lobe-1) ;
				lobeGauche -= getPixel(integral,pitch, x-(lobe+1)/2, y - lobe) ;
				lobeGauche += getPixel(integral,pitch, x-(lobe+1)/2 - lobe, y - lobe) ;
				lobeGauche -= getPixel(integral,pitch, x-(lobe+1)/2 - lobe, y + lobe-1) ;

				lobeCentre += getPixel(integral,pitch, x-(lobe+1)/2, y - lobe) ;
				lobeCentre -= getPixel(integral,pitch, x-(lobe+1)/2, y + lobe-1) ;
				lobeCentre += getPixel(integral,pitch, x+(lobe-1)/2, y + lobe-1) ;
				lobeCentre -= getPixel(integral,pitch, x+(lobe-1)/2, y - lobe) ;
				
				lobeDroit += getPixel(integral,pitch, x+(lobe-1)/2, y - lobe) ;
				lobeDroit -= getPixel(integral,pitch, x+(lobe-1)/2, y + lobe-1) ;
				lobeDroit += getPixel(integral,pitch, x+(lobe-1)/2 + lobe, y + lobe-1) ;
				lobeDroit -= getPixel(integral,pitch, x+(lobe-1)/2 + lobe, y - lobe) ;
				
				int dxx = lobeCentre - lobeDroit - lobeGauche ;
				
				// Derivee selon y
				int lobeHaut = 0, lobeBas = lobeCentre = 0 ;
				lobeHaut += getPixel(integral,pitch, x-lobe, y - (3*lobe +1)/2) ;
				lobeHaut -= getPixel(integral,pitch, x+lobe-1, y - (3*lobe +1)/2) ;
				lobeHaut += getPixel(integral,pitch, x+lobe-1, y - (lobe +1)/2) ;
				lobeHaut -= getPixel(integral,pitch, x-lobe, y - (lobe +1)/2) ;
				
				lobeCentre += getPixel(integral,pitch, x-lobe, y - (lobe +1)/2) ;
				lobeCentre -= getPixel(integral,pitch, x+lobe-1, y - (lobe +1)/2) ;
				lobeCentre += getPixel(integral,pitch, x+lobe-1, y + (lobe -1)/2) ;
				lobeCentre -= getPixel(integral,pitch, x-lobe, y + (lobe -1)/2) ;
				
				lobeBas += getPixel(integral,pitch, x-lobe, y + (lobe -1)/2) ;
				lobeBas -= getPixel(integral,pitch, x+lobe-1, y + (lobe -1)/2) ;
				lobeBas += getPixel(integral,pitch, x+lobe-1, y + (3*lobe -1)/2) ;
				lobeBas -= getPixel(integral,pitch, x-lobe, y + (3*lobe -1)/2) ;
				
				int dyy = lobeCentre - lobeHaut - lobeBas ;
				
				// Derivee selon xy
				int lobe00=0, lobe01=0, lobe10=0, lobe11=0;
				
				lobe00 += getPixel(integral,pitch, x-lobe-1, y-lobe -1) ;
				lobe00 -= getPixel(integral,pitch, x-1, y-lobe -1) ;
				lobe00 += getPixel(integral,pitch, x-1, y-1) ;
				lobe00 -= getPixel(integral,pitch, x-lobe-1, y-1) ;
				
				lobe01 += getPixel(integral,pitch, x, y-lobe-1) ;
				lobe01 -= getPixel(integral,pitch, x, y-1) ;
				lobe01 += getPixel(integral,pitch, x+lobe, y-1) ;
				lobe01 -= getPixel(integral,pitch, x+lobe, y-lobe-1) ;
				
				lobe10 += getPixel(integral,pitch, x-lobe-1, y) ;
				lobe10 -= getPixel(integral,pitch, x-1, y) ;
				lobe10 += getPixel(integral,pitch, x-1, y+lobe) ;
				lobe10 -= getPixel(integral,pitch, x-lobe-1, y+lobe) ;

				lobe11 += getPixel(integral,pitch, x, y) ;
				lobe11 -= getPixel(integral,pitch, x, y+lobe) ;
				lobe11 -= getPixel(integral,pitch, x+lobe, y) ;
				lobe11 += getPixel(integral,pitch, x+lobe, y+lobe) ;
				
				int dxy = lobe00 + lobe11 - lobe10 - lobe01 ;
				
				getPixel(out,pitch,x,y) = (int)((dxx*dyy- (0.9*dxy)*(0.9*dxy))/(area*area)) ;
  }
  else{
  	getPixel(out, pitch, x, y) =0;
  }
}

void CUDAcalculateGaussianDerivative(const IplImage* imageIntegrale, IplImage** out, int octave, int intervals){
	if(imageIntegrale->depth!=IPL_DEPTH_32S){
		std::cout << "Mauvais type d'images dans CUDAcalculateGaussianDerivative" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	uint *integralInCUDA, *a;
	uint pitch;
	//allocation de la memoire device
	if(hipSuccess != hipMallocPitch((void**)&integralInCUDA,&pitch,imageIntegrale->width*sizeof(unsigned int),imageIntegrale->height))
			std::cout << "erreur allocation" << std::endl;
	if(hipSuccess != hipMallocPitch((void**)&a,&pitch,imageIntegrale->width*sizeof(unsigned int),imageIntegrale->height))
			std::cout << "erreur allocation" << std::endl;
	
	clock_t timer=clock();
	//copie sur le device
	if(hipSuccess != hipMemcpy2D(a,pitch,imageIntegrale->imageData,imageIntegrale->widthStep,
			imageIntegrale->width*sizeof(unsigned int),imageIntegrale->height,hipMemcpyHostToDevice))
			std::cout << "erreur copie" << std::endl;
			
	std::cout << "CUDAcalculateGaussianDerivative : " << 1000*(clock()-timer)/CLOCKS_PER_SEC <<"ms"<< std::endl;
	
	// Calcul de la taille du filtre et des bordures
	int power = 1 ;
	for (int t=0 ; t<octave+1 ; t++)
	{
		power *= 2 ;
	}
	int borderSize = (3*(power*intervals + 1))/2+1  ;
	
	dim3 dimBlock( blocksize, blocksize );
    dim3 dimGrid( imageIntegrale->width/blocksize, imageIntegrale->height/blocksize);
	for (int inter=0 ; inter<intervals ; inter++){
		// Calcul de la surface pour normalisation d'echelle
		int lobe = power*(inter+1) + 1 ;
		int area = (3*lobe) * (3*lobe) ;
		//lancement des calculs
      	CUDAcalculateGaussianDerivative2<<<dimGrid, dimBlock>>>(integralInCUDA, a, imageIntegrale->width, imageIntegrale->height , pitch, lobe, area, borderSize);
	  	hipDeviceSynchronize();

		//recuperation depuis le device
		if(hipSuccess != hipMemcpy2D(out[inter]->imageData,out[inter]->widthStep,a,pitch,
			out[inter]->width*sizeof(unsigned int),out[inter]->height,hipMemcpyDeviceToHost))
			std::cout << "erreur copie" << std::endl;
	}
			
	std::cout << "CUDAcalculateGaussianDerivative : " << 1000*(clock()-timer)/(double)CLOCKS_PER_SEC <<"ms"<< std::endl;
	
	//liberation de la memoire du device
	hipFree(a);
	hipFree(integralInCUDA);
}
