#include "hip/hip_runtime.h"
#include "surfCUDA.h"
#include <cutil.h>
#include <stdlib.h>
#include <iostream>

#include <ctime>

const int blocksize = 10;

__global__ void CUDAcalculateGaussianDerivative2(uint* integral, uint** out, int width, int height, int pitch ,int intervals, int power, int borderSize) {
  int x = blockIdx.x * blockDim.x + threadIdx.x; 
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( x < width-borderSize && y < height-borderSize && x >= borderSize && y >= borderSize ){
  	for(int inter=0;inter<intervals;inter++){
  	int lobe = power*(inter+1) + 1 ;
	int area = (3*lobe) * (3*lobe) ;
  	// Derivee selon x
				int lobeGauche = 0, lobeCentre = 0, lobeDroit = 0 ;
				lobeGauche += unsignedGetPixel(integral,pitch, x-(lobe+1)/2, y + lobe-1) ;
				lobeGauche -= unsignedGetPixel(integral,pitch, x-(lobe+1)/2, y - lobe) ;
				lobeGauche += unsignedGetPixel(integral,pitch, x-(lobe+1)/2 - lobe, y - lobe) ;
				lobeGauche -= unsignedGetPixel(integral,pitch, x-(lobe+1)/2 - lobe, y + lobe-1) ;

				lobeCentre += unsignedGetPixel(integral,pitch, x-(lobe+1)/2, y - lobe) ;
				lobeCentre -= unsignedGetPixel(integral,pitch, x-(lobe+1)/2, y + lobe-1) ;
				lobeCentre += unsignedGetPixel(integral,pitch, x+(lobe-1)/2, y + lobe-1) ;
				lobeCentre -= unsignedGetPixel(integral,pitch, x+(lobe-1)/2, y - lobe) ;
				
				lobeDroit += unsignedGetPixel(integral,pitch, x+(lobe-1)/2, y - lobe) ;
				lobeDroit -= unsignedGetPixel(integral,pitch, x+(lobe-1)/2, y + lobe-1) ;
				lobeDroit += unsignedGetPixel(integral,pitch, x+(lobe-1)/2 + lobe, y + lobe-1) ;
				lobeDroit -= unsignedGetPixel(integral,pitch, x+(lobe-1)/2 + lobe, y - lobe) ;
				
				int dxx = lobeCentre - lobeDroit - lobeGauche ;
				
				// Derivee selon y
				int lobeHaut = 0, lobeBas = lobeCentre = 0 ;
				lobeHaut += unsignedGetPixel(integral,pitch, x-lobe, y - (3*lobe +1)/2) ;
				lobeHaut -= unsignedGetPixel(integral,pitch, x+lobe-1, y - (3*lobe +1)/2) ;
				lobeHaut += unsignedGetPixel(integral,pitch, x+lobe-1, y - (lobe +1)/2) ;
				lobeHaut -= unsignedGetPixel(integral,pitch, x-lobe, y - (lobe +1)/2) ;
				
				lobeCentre += unsignedGetPixel(integral,pitch, x-lobe, y - (lobe +1)/2) ;
				lobeCentre -= unsignedGetPixel(integral,pitch, x+lobe-1, y - (lobe +1)/2) ;
				lobeCentre += unsignedGetPixel(integral,pitch, x+lobe-1, y + (lobe -1)/2) ;
				lobeCentre -= unsignedGetPixel(integral,pitch, x-lobe, y + (lobe -1)/2) ;
				
				lobeBas += unsignedGetPixel(integral,pitch, x-lobe, y + (lobe -1)/2) ;
				lobeBas -= unsignedGetPixel(integral,pitch, x+lobe-1, y + (lobe -1)/2) ;
				lobeBas += unsignedGetPixel(integral,pitch, x+lobe-1, y + (3*lobe -1)/2) ;
				lobeBas -= unsignedGetPixel(integral,pitch, x-lobe, y + (3*lobe -1)/2) ;
				
				int dyy = lobeCentre - lobeHaut - lobeBas ;
				
				// Derivee selon xy
				int lobe00=0, lobe01=0, lobe10=0, lobe11=0;
				
				lobe00 += unsignedGetPixel(integral,pitch, x-lobe-1, y-lobe -1) ;
				lobe00 -= unsignedGetPixel(integral,pitch, x-1, y-lobe -1) ;
				lobe00 += unsignedGetPixel(integral,pitch, x-1, y-1) ;
				lobe00 -= unsignedGetPixel(integral,pitch, x-lobe-1, y-1) ;
				
				lobe01 += unsignedGetPixel(integral,pitch, x, y-lobe-1) ;
				lobe01 -= unsignedGetPixel(integral,pitch, x, y-1) ;
				lobe01 += unsignedGetPixel(integral,pitch, x+lobe, y-1) ;
				lobe01 -= unsignedGetPixel(integral,pitch, x+lobe, y-lobe-1) ;
				
				lobe10 += unsignedGetPixel(integral,pitch, x-lobe-1, y) ;
				lobe10 -= unsignedGetPixel(integral,pitch, x-1, y) ;
				lobe10 += unsignedGetPixel(integral,pitch, x-1, y+lobe) ;
				lobe10 -= unsignedGetPixel(integral,pitch, x-lobe-1, y+lobe) ;

				lobe11 += unsignedGetPixel(integral,pitch, x, y) ;
				lobe11 -= unsignedGetPixel(integral,pitch, x, y+lobe) ;
				lobe11 -= unsignedGetPixel(integral,pitch, x+lobe, y) ;
				lobe11 += unsignedGetPixel(integral,pitch, x+lobe, y+lobe) ;
				
				int dxy = lobe00 + lobe11 - lobe10 - lobe01 ;
				
				unsignedGetPixel(out[inter],pitch,x,y) = (int)((dxx*dyy- (0.9*dxy)*(0.9*dxy))/(area*area)) ;
  	}
  }
  else{
  	for(int inter=0;inter<intervals;inter++)
  		unsignedGetPixel(out[inter], pitch, x, y) =0;
  }
}

void CUDAcalculateGaussianDerivative(const IplImage* imageIntegrale, int octave, int intervals){
	if(imageIntegrale->depth!=IPL_DEPTH_32S){
		std::cout << "Mauvais type d'images dans CUDAcalculateGaussianDerivative" << std::endl;
		exit(EXIT_FAILURE);
	}
	

	//copie sur le device
	if(hipSuccess != hipMemcpy2D(CUDAintegral,CUDAintegralPitch,imageIntegrale->imageData,imageIntegrale->widthStep,
			imageIntegrale->width*sizeof(unsigned int),imageIntegrale->height,hipMemcpyHostToDevice))
			std::cout << "erreur copie" << std::endl;
	// Calcul de la taille du filtre et des bordures
	int power = 1 ;
	for (int t=0 ; t<octave+1 ; t++)
	{
		power *= 2 ;
	}
	int borderSize = (3*(power*intervals + 1))/2+1  ;

	
	dim3 dimBlock( blocksize, blocksize );
    dim3 dimGrid( imageIntegrale->width/blocksize, imageIntegrale->height/blocksize);
	//lancement des calculs
     CUDAcalculateGaussianDerivative2<<<dimGrid, dimBlock>>>(CUDAintegral, (uint**)CUDAadressImgs, imageIntegrale->width, imageIntegrale->height , CUDAimgsPitch, intervals, power , borderSize);

}

void CUDAretrieveGaussianDerivative(IplImage** out, int intervals){
	hipDeviceSynchronize();
	for (int inter=0 ; inter<intervals ; inter++){
		//recuperation depuis le device
		if(hipSuccess != hipMemcpy2D(out[inter]->imageData,out[inter]->widthStep,CUDAimgs[inter],CUDAimgsPitch,
			out[inter]->width*sizeof(unsigned int),out[inter]->height,hipMemcpyDeviceToHost))
			std::cout << "erreur récupération" << std::endl;
	}
}
