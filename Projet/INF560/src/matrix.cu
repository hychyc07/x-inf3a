#include "hip/hip_runtime.h"
#include <cutil.h>
#include <stdlib.h>
#include <stdio.h>

const int N = 2048; 
const int blocksize = 16;
const int MAX = 100;

__host__ void add_matrix_cpu(float* a, float *b, float *c, int N) {
  int i, j;
  for (i=0; i<N; i++)
    for (j=0; j<N; j++)
      c[i*N+j]=a[i*N+j]+b[i*N+j];
}

__global__ void add_matrix(float* a, float *b, float *c, int N ) {
  int i = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y; 
  int index = i + j*N; 
  if ( i < N && j < N )
    c[index] = a[index] + b[index];
}

void add_matrix(){
	
  int k;
  float *a = new float[N*N]; 
  float *b = new float[N*N]; 
  float *c = new float[N*N];
  unsigned int timer;
  unsigned int timer2;
  for ( int i = 0; i < N*N; ++i ) { 
    a[i] = 1.0f; b[i] = 3.5f; }
  float *ad, *bd, *cd; 
  const int size = N*N*sizeof(float); 
  hipMalloc( (void**)&ad, size ); 
  hipMalloc( (void**)&bd, size ); 
  hipMalloc( (void**)&cd, size );
  hipMemcpy( ad, a, size, hipMemcpyHostToDevice ); 
  hipMemcpy( bd, b, size, hipMemcpyHostToDevice );
  dim3 dimBlock( blocksize, blocksize ); 
  dim3 dimGrid( N/dimBlock.x, N/dimBlock.y ); 
  //CUT_SAFE_CALL(cutCreateTimer(&timer));
  //CUT_SAFE_CALL(cutStartTimer(timer));
  for (k=1;k<=MAX;k++) {
    add_matrix<<<dimGrid, dimBlock>>>( ad, bd, cd, N );
    hipDeviceSynchronize();
  }
  //CUT_SAFE_CALL(cutStopTimer(timer));
  //printf("Processing time on GPU: %f (ms)\n",cutGetTimerValue(timer)/MAX);
  //CUT_SAFE_CALL(cutDeleteTimer(timer));
  hipMemcpy( c, cd, size, hipMemcpyDeviceToHost );
  hipFree( ad ); 
  hipFree( bd ); 
  hipFree( cd ); 
  printf("Valeur: %f\n",c[0]);
  //CUT_SAFE_CALL(cutCreateTimer(&timer2));
  //CUT_SAFE_CALL(cutStartTimer(timer2));
  for (k=1;k<=MAX;k++)
    add_matrix_cpu(a,b,c, N);
  //CUT_SAFE_CALL(cutStopTimer(timer2));
  //printf("Processing time on CPU: %f (ms)\n",cutGetTimerValue(timer2)/MAX);
  //CUT_SAFE_CALL(cutDeleteTimer(timer2));  
  delete[] a; 
  delete[] b; 
  delete[] c;
}

/*int main() { 
	add_matrix();
  return EXIT_SUCCESS;
}*/