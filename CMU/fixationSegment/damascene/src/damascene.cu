
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <fcntl.h>
#include <float.h>
#include <unistd.h>
#include "texton.h"
#include "convert.h"
#include "intervening.h"
#include "lanczos.h"
#include "stencilMVM.h"

#include "localcues.h"
#include "combine.h"
#include "nonmax.h"
#include "spectralPb.h"
#include "globalPb.h"
#include "skeleton.h"

#define __TIMER_SPECFIC

#define TEXTON64 2
#define TEXTON32 1


int damascene(uint width, uint height, uint* imageData, float* gradData, float* oriData, bool gPb) {
  hipInit(0);
  chooseLargestGPU(true);

  int nEigNum = 9;
  float fEigTolerance = 1e-3;
  int nTextonChoice = TEXTON32;

  uint* devRgbU;
  int nPixels = width * height;
  hipMalloc((void**)(&devRgbU), nPixels*sizeof(uint));
  hipMemcpy(devRgbU, imageData, nPixels*sizeof(uint), hipMemcpyHostToDevice);

  uint timer;
#ifdef __TIMER_SPECFIC
  uint timer_specific;
#endif

  size_t totalMemory, availableMemory;
  hipMemGetInfo(&availableMemory,&totalMemory );
  printf("Available %zu bytes on GPU\n", availableMemory);

  cutCreateTimer(&timer);
  cutStartTimer(timer);
 
#ifdef __TIMER_SPECFIC
  cutCreateTimer(&timer_specific);
  cutStartTimer(timer_specific);
#endif

  float* devGreyscale;
  rgbUtoGreyF(width, height, devRgbU, &devGreyscale);

#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< rgbUtoGrayF | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif

  int* devTextons;
  findTextons(width, height, devGreyscale, &devTextons, nTextonChoice);

#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< texton | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif

  float* devL;
  float* devA;
  float* devB;
  rgbUtoLab3F(width, height, 2.5, devRgbU, &devL, &devA, &devB);

#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< rgbUtoLab3F | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif
  normalizeLab(width, height, devL, devA, devB);
#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< normalizeLab | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif
  int border = 30;
  int borderWidth = width + 2 * border;
  int borderHeight = height + 2 * border;
  float* devLMirrored;
  mirrorImage(width, height, border, devL, &devLMirrored);
/*   float* hostLMirrored = (float*)malloc(borderWidth * borderHeight * sizeof(float)); */
/*   hipMemcpy(hostLMirrored, devLMirrored, borderWidth * borderHeight * sizeof(float), hipMemcpyDeviceToHost); */
/*   writeFile("L.pb", borderWidth, borderHeight, hostLMirrored); */
 
  hipDeviceSynchronize();
  hipFree(devRgbU);
  hipFree(devGreyscale);
#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< mirrorImage | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif
  float* devBg;
  float* devCga;
  float* devCgb;
  float* devTg;
  int matrixPitchInFloats;
 
 uint localcuestimer; 
 cutCreateTimer(&localcuestimer);
 cutStartTimer(localcuestimer);

  localCues(width, height, devL, devA, devB, devTextons, &devBg, &devCga, &devCgb, &devTg, &matrixPitchInFloats, nTextonChoice);

  cutStopTimer(localcuestimer);
  printf("localcues time: %f seconds\n", cutGetTimerValue(localcuestimer)/1000.0);

#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< localcues | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif
   //float* hostG = (float*)malloc(sizeof(float) * nPixels); 
   //CUDA_SAFE_CALL(hipMemcpy(hostG, devBg, height*width*sizeof(float),hipMemcpyDeviceToHost));
   //cutSavePGMf("Bg.pgm", hostG, width, height);
   //free(hostG);

  hipFree(devTextons);
  hipFree(devL);
  hipFree(devA);
  hipFree(devB);
  

  float* devMPbO;
  float *devCombinedGradient;
  combine(width, height, matrixPitchInFloats, devBg, devCga, devCgb, devTg, &devMPbO, &devCombinedGradient, nTextonChoice);

#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< combine | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif

  CUDA_SAFE_CALL(hipFree(devBg));
  CUDA_SAFE_CALL(hipFree(devCga));
  CUDA_SAFE_CALL(hipFree(devCgb));
  CUDA_SAFE_CALL(hipFree(devTg));

  float* devMPb;
  hipMalloc((void**)&devMPb, sizeof(float) * nPixels);
  int* devOri;
  hipMalloc((void**)&devOri, sizeof(int) * nPixels);
  nonMaxSuppression(width, height, devMPbO, matrixPitchInFloats, devMPb, devOri);

  int* oriDataInd;
  oriDataInd=(int*)malloc(sizeof(int) * nPixels);
  hipMemcpy(oriDataInd, devOri, nPixels*sizeof(int), hipMemcpyDeviceToHost);
  for(int l=0;l<nPixels;l++){
    oriData[l]=oriDataInd[l]*M_PIl/8;
  }
  free(oriDataInd);
  CUDA_SAFE_CALL(hipFree(devOri));

#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< nonmaxsupression | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif

  if(!gPb){
    hipMemcpy(gradData, devMPb, nPixels*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_SAFE_CALL(hipFree(devMPb));
    return 0;
  }
  
  //int devMatrixPitch = matrixPitchInFloats * sizeof(float);
  int radius = 5;
  //int radius = 10;

  Stencil theStencil(radius, width, height, matrixPitchInFloats);
  int nDimension = theStencil.getStencilArea();
  float* devMatrix;
  intervene(theStencil, devMPb, &devMatrix);
  printf("Intervening contour completed\n");
 
#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< intervene | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif

  float* eigenvalues;
  float* devEigenvectors;
  //int nEigNum = 17;
  generalizedEigensolve(theStencil, devMatrix, matrixPitchInFloats, nEigNum, &eigenvalues, &devEigenvectors, fEigTolerance);

#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< generalizedEigensolve | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif
  float* devSPb = 0;
  size_t devSPb_pitch = 0;
  CUDA_SAFE_CALL(hipMallocPitch((void**)&devSPb, &devSPb_pitch, nPixels *  sizeof(float), 8));
  hipMemset(devSPb, 0, matrixPitchInFloats * sizeof(float) * 8);

  spectralPb(eigenvalues, devEigenvectors, width, height, nEigNum, devSPb, matrixPitchInFloats);

#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< spectralPb | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif
  float* devGPb = 0;
  CUDA_SAFE_CALL(hipMalloc((void**)&devGPb, sizeof(float) * nPixels));
  float* devGPball = 0;
  CUDA_SAFE_CALL(hipMalloc((void**)&devGPball, sizeof(float) * matrixPitchInFloats * 8));
  //StartCalcGPb(nPixels, matrixPitchInFloats, 8, devbg1, devbg2, devbg3, devcga1, devcga2, devcga3, devcgb1, devcgb2, devcgb3, devtg1, devtg2, devtg3, devSPb, devMPb, devGPball, devGPb);
  StartCalcGPb(nPixels, matrixPitchInFloats, 8, devCombinedGradient, devSPb, devMPb, devGPball, devGPb);
 
#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< StartCalcGpb | %f | ms\n", cutGetTimerValue(timer_specific));
  cutResetTimer(timer_specific);
  cutStartTimer(timer_specific);
#endif
  float* devGPb_thin = 0;
  CUDA_SAFE_CALL(hipMalloc((void**)&devGPb_thin, nPixels * sizeof(float) ));
  PostProcess(width, height, width, devGPb, devMPb, devGPb_thin); //note: 3rd param width is the actual pitch of the image
  NormalizeGpbAll(nPixels, 8, matrixPitchInFloats, devGPball);
  
  hipDeviceSynchronize();
  cutStopTimer(timer);
  printf("CUDA Status : %s\n", hipGetErrorString(hipGetLastError()));

#ifdef __TIMER_SPECFIC
  cutStopTimer(timer_specific);
  printf(">+< PostProcess | %f | ms\n", cutGetTimerValue(timer_specific));
#endif
  printf(">+< Computation time: | %f | seconds\n", cutGetTimerValue(timer)/1000.0);
  float* hostGPb = (float*)malloc(sizeof(float)*nPixels);
  memset(hostGPb, 0, sizeof(float) * nPixels);
  hipMemcpy(hostGPb, devGPb, sizeof(float)*nPixels, hipMemcpyDeviceToHost);

  /* thin image */
  hipMemcpy(gradData, devGPb_thin, sizeof(float)*nPixels, hipMemcpyDeviceToHost);
  /* end thin image */

  free(hostGPb);

  CUDA_SAFE_CALL(hipFree(devEigenvectors));
  CUDA_SAFE_CALL(hipFree(devCombinedGradient));
  CUDA_SAFE_CALL(hipFree(devSPb));
  CUDA_SAFE_CALL(hipFree(devGPb));
  CUDA_SAFE_CALL(hipFree(devGPb_thin));
  CUDA_SAFE_CALL(hipFree(devGPball));
}
